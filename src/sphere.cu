#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_world.h>

#include "raydata.h"

// Sphere variables
rtDeclareVariable(float3, center, , );
rtDeclareVariable(float, radius, , );

// The ray that will be intersected against
rtDeclareVariable(optix::Ray, theRay, rtCurrentRay, );
rtDeclareVariable(PerRayData, thePrd, rtPayload,  );

// The point and normal of intersection
//   the "attribute" qualifier is used to communicate between intersection and shading programs
//   These may only be written between rtPotentialIntersection and rtReportIntersection
rtDeclareVariable(float3, hitRecordP, attribute hitRecordP, );
rtDeclareVariable(float3, hitRecordNormal, attribute hitRecordNormal, );

inline __device__ float dot(float3 a, float3 b)
{
    return a.x*b.x + a.y*b.y + a.z*b.z;
}

// The sphere bounding box program
//   The pid parameter enables specifying a primitive withing this geometry
//   since there is only 1 primitive (the sphere), the pid is ignored here
RT_PROGRAM void getBounds(int pid, float result[6])
{
    optix::Aabb* aabb = (optix::Aabb*)result;
    aabb->m_min = center - radius;
    aabb->m_max = center + radius;
}


// The sphere intersection program
//   this function calls rtReportIntersection if an intersection occurs
//   As above, pid refers to a specific primitive, is ignored
RT_PROGRAM void intersection(int pid)
{
    float3 oc = theRay.origin - center;
    float a = dot(theRay.direction, theRay.direction);
    float b = dot(oc, theRay.direction);
    float c = dot(oc, oc) - radius*radius;
    float discriminant = b*b - a*c;

    if (discriminant < 0.0f) return;
        
    float t = (-b - sqrtf(discriminant)) / a;
    if (rtPotentialIntersection(t))
    {
        hitRecordP = theRay.origin + t * theRay.direction;
        hitRecordNormal = (hitRecordP - center) / radius;
        rtReportIntersection(0);
    }
    t = (b - sqrtf(discriminant)) / a;
    if (rtPotentialIntersection(t))
    {
        hitRecordP = theRay.origin + t * theRay.direction;
        hitRecordNormal = (hitRecordP - center) / radius;
        rtReportIntersection(0);
    }
}


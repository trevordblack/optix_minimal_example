#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "raydata.h"

using namespace optix;

// Optix program built-in indices
rtDeclareVariable(uint2, theLaunchIndex, rtLaunchIndex, );
rtDeclareVariable(uint2, theLaunchDim, rtLaunchDim, );

// Ray state variables
rtDeclareVariable(optix::Ray, theRay, rtCurrentRay, );
rtDeclareVariable(PerRayData, thePrd, rtPayload,  );

// "Global" variables
rtDeclareVariable(rtObject, sysWorld, , );

rtBuffer<float3, 2> sysOutputBuffer;

RT_PROGRAM void rayGenProgram()
{
    float3 lowerLeftCorner = make_float3(-2.0f, -1.0f, -1.0f);
    float3 horizontal = make_float3(4.0f, 0.0f, 0.0f);
    float3 vertical = make_float3(0.0f, 2.0f, 0.0f);
    float3 origin = make_float3(0.0f, 0.0f, 0.0f);

    float u = float(theLaunchIndex.x) / float(theLaunchDim.x);
    float v = float(theLaunchIndex.y) / float(theLaunchDim.y);

    float3 direction = lowerLeftCorner + (u*horizontal) + (v*vertical) - origin; 

    optix::Ray theRay = optix::make_Ray( 
            origin,        // origin
            direction,     // direction
            0,             // raytype
            0.000001f,     // tmin (epsilon)
            RT_DEFAULT_MAX // tmax
    );

    PerRayData thePrd;
    rtTrace(sysWorld, theRay, thePrd);

    float3 drawColor = make_float3(0.0f, 0.0f, 0.0f);

    if (thePrd.scatterEvent == miss)
    { // Didn't hit anything
        float3 unitDirection = normalize(direction);
        float t = 0.5f * (unitDirection.y + 1.0f);
        drawColor = (1.0f-t) * make_float3(1.0f, 1.0f, 1.0f)
                        + t * make_float3(0.5f, 0.7f, 1.0f);
    }
    else
    { // hit something
        drawColor = 0.5f * (thePrd.attenuation + make_float3(1.0f,1.0f,1.0f));
    }

    sysOutputBuffer[theLaunchIndex] = drawColor;
}

